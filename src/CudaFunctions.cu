#include "hip/hip_runtime.h"
#include "../include/CudaFunctions.h"

__device__
void computeSadOverBlockCuda(
        int minYL,
        int minXL,
        int minYR,
        int minXR,
        int width,
        int height,
        int imageWidth,
        const uint8_t* leftImageData,
        const uint8_t* rightImageData,
        int* sum) {

    *sum = 0;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            // __usad(a, b, c) = |a-b| + c
            *sum += __usad(
                    leftImageData[((y + minYL) * imageWidth) + (x + minXL)],
                    rightImageData[((y + minYR) * imageWidth) + (x + minXR)],
                    0);
        }
    }
}

__device__
void computeDisparityForPixelCuda(
        int y, 
        int x,
        int imageWidth,
        int imageHeight,
        int blockSize,
        int leftScanSteps,
        int rightScanSteps,
        const uint8_t* leftImageData,
        const uint8_t* rightImageData,
        float* output) {

    float disparityBuf[512];
    int maxBlockStep = (blockSize - 1) / 2;

    int templateLeftHalfWidth = min(x, maxBlockStep);
    int templateRightHalfWidth = min(imageWidth - x - 1, maxBlockStep);
    int templateTopHalfHeight = min(y, maxBlockStep);
    int templateBottomHalfHeight = min(imageHeight - y - 1, maxBlockStep);

    int templateWidth = templateLeftHalfWidth + templateRightHalfWidth + 1;
    int templateHeight = templateTopHalfHeight + templateBottomHalfHeight + 1;

    int leftMinY = y - templateTopHalfHeight;
    int leftMinX = x - templateLeftHalfWidth;

    int rightMinStartX = max(0, x - leftScanSteps - templateLeftHalfWidth);
    int rightMaxStartX = min(imageWidth - templateWidth, x + rightScanSteps - templateLeftHalfWidth);

    int numSteps = rightMaxStartX - rightMinStartX;

    int bestIndex = 0;
    int bestSadValue = 2147483646; // value of std::numeric_limits<int>::max() - 1
    int zeroDisparityIndex = x - rightMinStartX - templateLeftHalfWidth;

    for (int xx = rightMinStartX; xx <= rightMaxStartX; xx++) {
        int sad = 0;
        computeSadOverBlockCuda(
            leftMinY,
            leftMinX,
            leftMinY, // Ys are aligned for the two images
            xx,
            templateWidth,
            templateHeight,
            imageWidth,
            leftImageData, 
            rightImageData,
            &sad);

        disparityBuf[xx - rightMinStartX] = sad;

        if (sad < bestSadValue) {
            bestSadValue = sad;
            bestIndex = xx - rightMinStartX;
        }
    }

    float disparity = __int2float_rn(abs(bestIndex - zeroDisparityIndex));
    if ((bestIndex == 0)
        ||
        (bestIndex == numSteps)
        ||
        (bestSadValue == 0)) {
        *output = disparity;
    } else { 
        float c3 = disparityBuf[bestIndex+1];
        float c2 = disparityBuf[bestIndex];
        float c1 = disparityBuf[bestIndex-1];

        *output = disparity - (0.5 * ((c3 - c1) / (c1 - (2*c2) + c3)));
    }
}

__global__ 
void computeDisparityCudaInternal(
        int height,
        int width,
        int blockSize,
        int leftScanSteps,
        int rightScanSteps,
        uint8_t* leftImageData,
        uint8_t* rightImageData,
        float* disparityData) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int numElements = height * width;
    for (int i = index; i < numElements; i += stride) {
        int y = i / width;
        int x = i % width;

        computeDisparityForPixelCuda(
            y,
            x,
            width,
            height,
            blockSize,
            leftScanSteps,
            rightScanSteps,
            leftImageData,
            rightImageData,
            disparityData + i);
    }
}

static uint8_t* leftCudaData = NULL;
static uint8_t* rightCudaData = NULL;
static float* disparityCudaData = NULL;

void destroyCudaMemoryBuffers() {
    if (leftCudaData != NULL) {
        hipFree(leftCudaData);
        leftCudaData = NULL;
    }

    if (rightCudaData != NULL) {
        hipFree(rightCudaData);
        rightCudaData = NULL;
    }

    if (disparityCudaData != NULL) {
        hipFree(disparityCudaData);
        disparityCudaData = NULL;
    }
}

void computeDisparityCuda(
        int imageHeight,
        int imageWidth, 
        int blockSize,
        int leftScanSteps,
        int rightScanSteps,
        uint8_t* leftImageData,
        uint8_t* rightImageData,
        float* disparityData) {

    int numElements = imageHeight * imageWidth;

    if (leftCudaData == NULL) {
        hipMallocManaged(&leftCudaData, numElements * sizeof(uint8_t));
        hipMallocManaged(&rightCudaData, numElements * sizeof(uint8_t));
        hipMallocManaged(&disparityCudaData, numElements * sizeof(float));
    }

    hipMemcpy(leftCudaData, leftImageData, numElements * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(rightCudaData, rightImageData, numElements * sizeof(uint8_t), hipMemcpyHostToDevice);

    int numThreads = 256;
    int numBlocks = ceil(((float)numElements) / ((float)numThreads));
    computeDisparityCudaInternal<<<numBlocks, numThreads>>>(
        imageHeight,
        imageWidth,
        blockSize,
        leftScanSteps,
        rightScanSteps,
        leftCudaData,
        rightCudaData,
        disparityCudaData);

    hipDeviceSynchronize();

    hipMemcpy(disparityData, disparityCudaData, numElements * sizeof(float), hipMemcpyDeviceToHost);
}
