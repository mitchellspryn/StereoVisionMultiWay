#include "hip/hip_runtime.h"
#include "../include/CudaSimdFunctions.h"

__device__
void computeSadOverBlockCudaSimd(
        int minYL,
        int minXL,
        int minYR,
        int minXR,
        int width,
        int height,
        int imageWidth,
        const uint8_t* leftImageData,
        const uint8_t* rightImageData,
        int* sum) {
    *sum = 0;
    int numStrides = width / 4;
    for (int y = 0; y < height; y++) {
        for (int n = 0; n < numStrides; n++) {
            int  leftBaseIdx = ((y+minYL)*imageWidth) + minXL + (n*4);
            int rightBaseIdx = ((y+minYR)*imageWidth) + minXR + (n*4);

            uint32_t leftVal = 
                (leftImageData[leftBaseIdx+3] << 24)
                |
                (leftImageData[leftBaseIdx+2] << 16)
                |
                (leftImageData[leftBaseIdx+1] << 8)
                |
                (leftImageData[leftBaseIdx+0]);

            uint32_t rightVal = 
                (rightImageData[rightBaseIdx+3] << 24)
                |
                (rightImageData[rightBaseIdx+2] << 16)
                |
                (rightImageData[rightBaseIdx+1] << 8)
                |
                (rightImageData[rightBaseIdx+0]);

            *sum += __vsadu4(leftVal, rightVal);
        }

        for (int x = numStrides*4; x < width; x++) {
             //__usad(a, b, c) = |a-b| + c
            *sum += __usad(
                    leftImageData[((y + minYL) * imageWidth) + (x + minXL)],
                    rightImageData[((y + minYR) * imageWidth) + (x + minXR)],
                    0);
        }
    }
}

__device__
void computeDisparityForPixelCudaSimd(
        int y, 
        int x,
        int imageWidth,
        int imageHeight,
        int blockSize,
        int leftScanSteps,
        int rightScanSteps,
        const uint8_t* leftImageData,
        const uint8_t* rightImageData,
        float* output) {

    float disparityBuf[512];
    int maxBlockStep = (blockSize - 1) / 2;

    int templateLeftHalfWidth = min(x, maxBlockStep);
    int templateRightHalfWidth = min(imageWidth - x - 1, maxBlockStep);
    int templateTopHalfHeight = min(y, maxBlockStep);
    int templateBottomHalfHeight = min(imageHeight - y - 1, maxBlockStep);

    int templateWidth = templateLeftHalfWidth + templateRightHalfWidth + 1;
    int templateHeight = templateTopHalfHeight + templateBottomHalfHeight + 1;

    int leftMinY = y - templateTopHalfHeight;
    int leftMinX = x - templateLeftHalfWidth;

    int rightMinStartX = max(0, x - leftScanSteps - templateLeftHalfWidth);
    int rightMaxStartX = min(imageWidth - templateWidth, x + rightScanSteps - templateLeftHalfWidth);

    int numSteps = rightMaxStartX - rightMinStartX;

    int bestIndex = 0;
    int bestSadValue = 2147483646; // value of std::numeric_limits<int>::max() - 1
    int zeroDisparityIndex = x - rightMinStartX - templateLeftHalfWidth;

    for (int xx = rightMinStartX; xx <= rightMaxStartX; xx++) {
        int sad = 0;
        computeSadOverBlockCudaSimd(
            leftMinY,
            leftMinX,
            leftMinY, // Ys are aligned for the two images
            xx,
            templateWidth,
            templateHeight,
            imageWidth,
            leftImageData, 
            rightImageData,
            &sad);

        disparityBuf[xx - rightMinStartX] = sad;

        if (sad < bestSadValue) {
            bestSadValue = sad;
            bestIndex = xx - rightMinStartX;
        }
    }

    float disparity = __int2float_rn(abs(bestIndex - zeroDisparityIndex));
    if ((bestIndex == 0)
        ||
        (bestIndex == numSteps)
        ||
        (bestSadValue == 0)) {
        *output = disparity;
    } else { 
        float c3 = disparityBuf[bestIndex+1];
        float c2 = disparityBuf[bestIndex];
        float c1 = disparityBuf[bestIndex-1];

        *output = disparity - (0.5 * ((c3 - c1) / (c1 - (2*c2) + c3)));
    }
}

__global__ 
void computeDisparityCudaInternalSimd(
        int height,
        int width,
        int blockSize,
        int leftScanSteps,
        int rightScanSteps,
        uint8_t* leftImageData,
        uint8_t* rightImageData,
        float* disparityData) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int numElements = height * width;
    for (int i = index; i < numElements; i += stride) {
        int y = i / width;
        int x = i % width;

        computeDisparityForPixelCudaSimd(
            y,
            x,
            width,
            height,
            blockSize,
            leftScanSteps,
            rightScanSteps,
            leftImageData,
            rightImageData,
            disparityData + i);
    }
}

static uint8_t* leftCudaData = NULL;
static uint8_t* rightCudaData = NULL;
static float* disparityCudaData = NULL;

void destroyCudaMemoryBuffersSimd() {
    if (leftCudaData != NULL) {
        hipFree(leftCudaData);
        leftCudaData = NULL;
    }

    if (rightCudaData != NULL) {
        hipFree(rightCudaData);
        rightCudaData = NULL;
    }

    if (disparityCudaData != NULL) {
        hipFree(disparityCudaData);
        disparityCudaData = NULL;
    }
}

void computeDisparityCudaSimd(
        int imageHeight,
        int imageWidth, 
        int blockSize,
        int leftScanSteps,
        int rightScanSteps,
        uint8_t* leftImageData,
        uint8_t* rightImageData,
        float* disparityData) {

    int numElements = imageHeight * imageWidth;

    if (leftCudaData == NULL) {
        hipMallocManaged(&leftCudaData, numElements * sizeof(uint8_t));
        hipMallocManaged(&rightCudaData, numElements * sizeof(uint8_t));
        hipMallocManaged(&disparityCudaData, numElements * sizeof(float));
    }

    hipMemcpy(leftCudaData, leftImageData, numElements * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(rightCudaData, rightImageData, numElements * sizeof(uint8_t), hipMemcpyHostToDevice);

    int numThreads = 256;
    int numBlocks = ceil(((float)numElements) / ((float)numThreads));
    computeDisparityCudaInternalSimd<<<numBlocks, numThreads>>>(
        imageHeight,
        imageWidth,
        blockSize,
        leftScanSteps,
        rightScanSteps,
        leftCudaData,
        rightCudaData,
        disparityCudaData);

    hipDeviceSynchronize();

    hipMemcpy(disparityData, disparityCudaData, numElements * sizeof(float), hipMemcpyDeviceToHost);
}
